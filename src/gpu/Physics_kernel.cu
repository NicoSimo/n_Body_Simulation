#include "hip/hip_runtime.h"
#ifndef DIRECT_SUM_KERNEL_H_
#define DIRECT_SUM_KERNEL_H_

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>
#include <glm/glm.hpp>
#include "constants.h"
#include "err.h"
#include "Body.h"

#define BLOCK_SIZE 256

__device__ float getDistance(glm::vec2 pos1, glm::vec2 pos2)
{
    return glm::length(pos1 - pos2);
}

__device__ bool isCollide(Body &b1, Body &b2)
{
    return b1.radius + b2.radius + COLLISION_TH > getDistance(glm::vec2(b1.position.x, b1.position.y), glm::vec2(b2.position.x, b2.position.y));
}

__global__ void DirectSumTiledKernel(Body *bodies, int n)
{
    __shared__ Body Bds[BLOCK_SIZE];

    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int i = bx * blockDim.x + tx;

    if (i < n)
    {
        Body &bi = bodies[i];
        glm::vec2 forceAccum = glm::vec2(0.0f, 0.0f);
        bi.acceleration = glm::vec3(0.0f, 0.0f, 0.0f); // Reset acceleration
        for (int tile = 0; tile < gridDim.x; ++tile)
        {
            Bds[tx] = bodies[tile * blockDim.x + tx];
            __syncthreads();

            for (int b = 0; b < BLOCK_SIZE; ++b)
            {
                int j = tile * blockDim.x + b;
                if (j < n)
                {
                    Body bj = Bds[b];
                    if (!isCollide(bi, bj) && bi.isDynamic)
                    {
                        glm::vec2 rij = glm::vec2(bj.position.x - bi.position.x, bj.position.y - bi.position.y);
                        float r = glm::length(rij) + E;
                        float f = (GRAVITY * bi.mass * bj.mass) / (r * r * r + (E * E));
                        glm::vec2 force = rij * f;
                        forceAccum += force / bi.mass;
                    }
                }
            }
            __syncthreads();
        }
        bi.acceleration += glm::vec3(forceAccum, 0.0f);
        bi.velocity += bi.acceleration * DT;
        bi.position += bi.velocity * DT;
    }
}

Body *initRandomBodies(int n)
{
    Body *bodies = new Body[n];
    srand(time(NULL));
    float maxDistance = MAX_DIST;
    float minDistance = MIN_DIST;
    glm::vec3 centerPos = {CENTERX, CENTERY, 0.0f};
    for (int i = 0; i < n - 1; ++i)
    {
        float angle = 2 * M_PI * (rand() / (float)RAND_MAX);
        float radius = (maxDistance - minDistance) * (rand() / (float)RAND_MAX) + minDistance;

        float x = centerPos.x + radius * cos(angle);
        float y = centerPos.y + radius * sin(angle);
        glm::vec3 position = {x, y, 0.0f};
        bodies[i] = Body(EARTH_MASS, EARTH_DIA, position, glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), true);
    }

    bodies[n - 1] = Body(SUN_MASS, SUN_DIA, centerPos, glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), false);
    return bodies;
}


Body *Simulate(int n)
{

    Body *bodies = new Body[n];
    srand(time(NULL));
    float maxDistance = MAX_DIST;
    float minDistance = MIN_DIST;
    glm::vec2 centerPos = {CENTERX, CENTERY};
    for (int i = 0; i < n - 1; ++i)
    {

        float angle = 2 * M_PI * (rand() / (float)RAND_MAX);
        // Generate random distance from center within the given max distance
        float radius = (maxDistance - minDistance) * (rand() / (float)RAND_MAX) + minDistance;

        // Calculate coordinates of the point
        float x = centerPos.x + radius * std::cos(angle);
        float y = centerPos.y + radius * std::sin(angle);

        glm::vec2 position = {x, y};

        float distance = sqrt(pow(x - centerPos.x, 2) + pow(y - centerPos.y, 2));
        glm::vec2 r = {position.x - centerPos.x, position.y - centerPos.y};
        glm::vec2 a = {r.x / distance, r.y / distance};

        // Calculate velocity vector components
        float esc = sqrt((GRAVITY * SUN_MASS) / (distance));
        glm::vec2 velocity = {-a.y * esc, a.x * esc};

        bodies[i].isDynamic = true;
        bodies[i].mass = EARTH_MASS;
        bodies[i].radius = EARTH_DIA;
        bodies[i].position = position;
        bodies[i].velocity = velocity;
        bodies[i].acceleration = {0.0, 0.0};
    }

    bodies[n - 1].isDynamic = false;
    bodies[n - 1].mass = SUN_MASS;
    bodies[n - 1].radius = SUN_DIA;
    bodies[n - 1].position = centerPos;
    bodies[n - 1].velocity = {0.0, 0.0};
    bodies[n - 1].acceleration = {0.0, 0.0};
    return bodies;
}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        std::cerr << "Usage: " << argv[0] << " <number_of_bodies> <2d_or_3d>" << std::endl;
        return 1;
    }

    int nBodies = std::atoi(argv[1]);
    if (nBodies <= 0)
    {
        std::cerr << "Error, number of bodies must be a positive integer" << std::endl;
        return 1;
    }
    
    h_bodies = initSpiralBodies(nBodies);
    
    int bytes = nBodies * sizeof(Body);

    Body *d_bodies;
    
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_bodies, bytes));
    CHECK_CUDA_ERROR(hipMemcpy(d_bodies, h_bodies, bytes, hipMemcpyHostToDevice));

    int blockSize = BLOCK_SIZE;
    int gridSize = ceil((float)nBodies / blockSize);
    int it = 0;
    int iters = 1000;

    while (it < iters)
    {
        DirectSumTiledKernel<<<gridSize, blockSize>>>(d_bodies, nBodies);
        CHECK_LAST_CUDA_ERROR();
        CHECK_CUDA_ERROR(hipMemcpy(h_bodies, d_bodies, bytes, hipMemcpyHostToDevice));
        // Visualization or logging logic can go here if needed
        ++it;
    }

    CHECK_CUDA_ERROR(hipFree(d_bodies));
    free(h_bodies);

    CHECK_LAST_CUDA_ERROR();
    return 0;
}

#endif
